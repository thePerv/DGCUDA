
#include <hip/hip_runtime.h>
/* 2dadvec_kernels_euler.cu
 *
 * This file contains the kernels for the 2D euler DG method.
 *
 * d_t [   rho   ] + d_x [     rho * u    ] + d_y [    rho * v     ] = 0
 * d_t [ rho * u ] + d_x [ rho * u^2 + p  ] + d_y [   rho * u * v  ] = 0
 * d_t [ rho * v ] + d_x [  rho * u * v   ] + d_y [  rho * v^2 + p ] = 0
 * d_t [    E    ] + d_x [ u * ( E +  p ) ] + d_y [ v * ( E +  p ) ] = 0
 *

 */

#define PI 3.14159
#define GAMMA 1.4

/***********************
 *
 * DEVICE VARIABLES
 *
 ***********************/
/* These are always prefixed with d_ for "device" */
double *d_c;                 // coefficients for [rho, rho * u, rho * v, E]
double *d_quad_rhs;          // the right hand side containing the quadrature contributions
double *d_left_riemann_rhs;  // the right hand side containing the left riemann contributions
double *d_right_riemann_rhs; // the right hand side containing the right riemann contributions

// TODO: switch to low storage runge-kutta
// runge kutta variables
double *d_kstar;
double *d_k1;
double *d_k2;
double *d_k3;
double *d_k4;

// precomputed basis functions 
// TODO: maybe making these 2^n makes sure the offsets are cached more efficiently? who knows...
// precomputed basis functions ordered like so
//
// [phi_1(r1, s1), phi_1(r2, s2), ... , phi_1(r_nq, s_nq)   ]
// [phi_2(r1, s1), phi_2(r2, s2), ... , phi_2(r_nq, s_nq)   ]
// [   .               .           .            .           ]
// [   .               .           .            .           ]
// [   .               .           .            .           ]
// [phi_np(r1, s1), phi_np(r2, s2), ... , phi_np(r_nq, s_nq)]
//
__device__ __constant__ double basis[2048];
// note: these are multiplied by the weights
__device__ __constant__ double basis_grad_x[2048]; 
__device__ __constant__ double basis_grad_y[2048]; 

// precomputed basis functions evaluated along the sides. ordered
// similarly to basis and basis_grad_{x,y} but with one "matrix" for each side
// starting with side 0. to get to each side, offset with:
//      side_number * n_p * num_quad1d.
__device__ __constant__ double basis_side[1024];
__device__ __constant__ double basis_vertex[256];

// weights for 2d and 1d quadrature rules
__device__ __constant__ double w[32];
__device__ __constant__ double w_oned[16];

__device__ __constant__ double r1[32];
__device__ __constant__ double r2[32];
__device__ __constant__ double r_oned[32];

void set_basis(void *value, int size) {
    hipMemcpyToSymbol(HIP_SYMBOL("basis"), value, size * sizeof(double));
}
void set_basis_grad_x(void *value, int size) {
    hipMemcpyToSymbol(HIP_SYMBOL("basis_grad_x"), value, size * sizeof(double));
}
void set_basis_grad_y(void *value, int size) {
    hipMemcpyToSymbol(HIP_SYMBOL("basis_grad_y"), value, size * sizeof(double));
}
void set_basis_side(void *value, int size) {
    hipMemcpyToSymbol(HIP_SYMBOL("basis_side"), value, size * sizeof(double));
}
void set_basis_vertex(void *value, int size) {
    hipMemcpyToSymbol(HIP_SYMBOL("basis_vertex"), value, size * sizeof(double));
}
void set_w(void *value, int size) {
    hipMemcpyToSymbol(HIP_SYMBOL("w"), value, size * sizeof(double));
}
void set_w_oned(void *value, int size) {
    hipMemcpyToSymbol(HIP_SYMBOL("w_oned"), value, size * sizeof(double));
}
void set_r1(void *value, int size) {
    hipMemcpyToSymbol(HIP_SYMBOL("r1"), value, size * sizeof(double));
}
void set_r2(void *value, int size) {
    hipMemcpyToSymbol(HIP_SYMBOL("r2"), value, size * sizeof(double));
}
void set_r_oned(void *value, int size) {
    hipMemcpyToSymbol(HIP_SYMBOL("r_oned"), value, size * sizeof(double));
}

// tells which side (1, 2, or 3) to evaluate this boundary integral over
int *d_left_side_number;
int *d_right_side_number;

double *d_J;        // jacobian determinant 
double *d_min_J;      // for the min sized jacobian
double *d_s_length; // length of sides

// the num_elem values of the x and y coordinates for the two vertices defining a side
// TODO: can i delete these after the lengths are precomputed?
//       maybe these should be in texture memory?
double *d_s_V1x;
double *d_s_V1y;
double *d_s_V2x;
double *d_s_V2y;

// the num_elem values of the x and y partials
double *d_xr;
double *d_yr;
double *d_xs;
double *d_ys;

// the K indices of the sides for each element ranged 0->H-1
int *d_elem_s1;
int *d_elem_s2;
int *d_elem_s3;

// vertex x and y coordinates on the mesh which define an element
// TODO: can i delete these after the jacobians are precomputed?
//       maybe these should be in texture memory?
double *d_V1x;
double *d_V1y;
double *d_V2x;
double *d_V2y;
double *d_V3x;
double *d_V3y;

// stores computed values at three vertices
double *d_Uv1;
double *d_Uv2;
double *d_Uv3;

// normal vectors for the sides
double *d_Nx;
double *d_Ny;

// index lists for sides
int *d_left_elem;  // index of left  element for side idx
int *d_right_elem; // index of right element for side idx

/***********************
 *
 * DEVICE FUNCTIONS
 *
 ***********************/
/* riemann solver
 *
 * evaluates the riemann problem over the boundary using Gaussian quadrature
 * with Legendre polynomials as basis functions.
 */
__device__ double riemann(double u_left, double u_right) {
    return 0.5 * (u_left + u_right);
}

__device__ double pressure(double rho, double u, double v, double E) {
    return (GAMMA - 1) * (E - (u*u + v*v) / 2 * rho);
}

/***********************
 *
 * INITIAL CONDITIONS
 *
 ***********************/

/* initial condition function
 *
 * returns the value of the intial condition at point x
 */
__device__ double rho0(double x, double y) {
    return 2. + x ;
}
__device__ double u0(double x, double y) {
    return 1.;
}
__device__ double v0(double x, double y) {
    return 1.;
}
__device__ double E0(double x, double y) {
    return 1. / (GAMMA - 1) + (powf(u0(x, y), 2) + powf(v0(x, y), 2)) / 2. * rho0(x, y);
}

/* boundary exact
 *
 * returns the exact boundary conditions
 */
__device__ double boundary_exact_rho(double x, double y, double t) {
    return rho0(x, y);
}
__device__ double boundary_exact_u(double x, double y, double t) {
    return u0(x, y);
}
__device__ double boundary_exact_v(double x, double y, double t) {
    return v0(x, y);
}
__device__ double boundary_exact_E(double x, double y, double t) {
    return E0(x, y);
}

/* u exact
 *
 * returns the exact value of u for error measurement.
 */
__device__ double uexact(double x, double y, double t) {
    return u0(x, y);
}

/* initial conditions
 *
 * computes the coefficients for the initial conditions
 * THREADS: num_elem
 */
__global__ void init_conditions(double *c, double *J,
                                double *V1x, double *V1y,
                                double *V2x, double *V2y,
                                double *V3x, double *V3y,
                                int n_quad, int n_p, int num_elem) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int i, j;
    double x, y, rho, u, v, E;

    if (idx < num_elem) {
        for (i = 0; i < n_p; i++) {
            rho = 0.;
            u = 0.;
            v = 0.;
            E = 0.;
            // perform quadrature
            for (j = 0; j < n_quad; j++) {
                // map from the canonical element to the actual point on the mesh
                // x = x2 * r + x3 * s + x1 * (1 - r - s)
                x = r1[j] * V2x[idx] + r2[j] * V3x[idx] + (1 - r1[j] - r2[j]) * V1x[idx];
                y = r1[j] * V2y[idx] + r2[j] * V3y[idx] + (1 - r1[j] - r2[j]) * V1y[idx];

                // evaluate rho, u, v, E there
                rho += w[j] * rho0(x, y) * basis[i * n_quad + j];
                u   += w[j] * u0(x, y) * rho0(x, y) * basis[i * n_quad + j];
                v   += w[j] * v0(x, y) * rho0(x, y) * basis[i * n_quad + j];
                E   += w[j] * E0(x, y) * basis[i * n_quad + j];
            }

            c[num_elem * n_p * 0 + i * num_elem + idx] = rho;
            c[num_elem * n_p * 1 + i * num_elem + idx] = u; // we actually calculate and store rho * u
            c[num_elem * n_p * 2 + i * num_elem + idx] = v; // we actually calculate and store rho * v
            c[num_elem * n_p * 3 + i * num_elem + idx] = E;
        } 
    }
}

/* find min jacobian
 *
 * returns the min jacobian inside of min_J. 
 * each block computes the min jacobian inside of that block and stores it in the
 * blockIdx.x spot of the shared min_J variable.
 * NOTE: this is fixed for 256 threads.
 */
__global__ void min_jacobian(double *J, double *min_J, int num_elem) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int tid = threadIdx.x;
    int i   = (blockIdx.x * 256 * 2) + threadIdx.x;

    __shared__ double s_min[256];

    if (idx < num_elem) {
        // set all of min to J[idx] initially
        s_min[tid] = J[idx];
        __syncthreads();

        // test a few
        while (i < num_elem) {
            s_min[tid] = (s_min[tid] < J[i]) ? s_min[tid] : J[i];
            s_min[tid] = (s_min[tid] < J[i + 256]) ? s_min[tid] : J[i];
            i += gridDim.x * 256 * 2;
            __syncthreads();
        }

        // first half of the warps
        __syncthreads();
        if (tid < 128) {
            s_min[tid] = (s_min[tid] < s_min[tid + 128]) ? s_min[tid] : s_min[tid + 128];
        }

        // first and second warps
        __syncthreads();
        if (tid < 64) {
            s_min[tid] = (s_min[tid] < s_min[tid + 64]) ? s_min[tid] : s_min[tid + 64];
        }

        // unroll last warp
        __syncthreads();
        if (tid < 32) {
            if (blockDim.x >= 64) {
                s_min[tid] = (s_min[tid] < s_min[tid + 32]) ? s_min[tid] : s_min[tid + 32];
            }
            if (blockDim.x >= 32) {
                s_min[tid] = (s_min[tid] < s_min[tid + 16]) ? s_min[tid] : s_min[tid + 16];
            }
            if (blockDim.x >= 16) {
                s_min[tid] = (s_min[tid] < s_min[tid + 8]) ? s_min[tid] : s_min[tid + 8];
            }
            if (blockDim.x >= 8) {
                s_min[tid] = (s_min[tid] < s_min[tid + 4]) ? s_min[tid] : s_min[tid + 4];
            }
            if (blockDim.x >= 4) {
                s_min[tid] = (s_min[tid] < s_min[tid + 2]) ? s_min[tid] : s_min[tid + 2];
            }
            if (blockDim.x >= 2) {
                s_min[tid] = (s_min[tid] < s_min[tid + 1]) ? s_min[tid] : s_min[tid + 1];
            }
        }

        __syncthreads();
        if (tid == 0) {
            min_J[blockIdx.x] = s_min[0];
        }
    }
}

/***********************
 *
 * PRECOMPUTING
 *
 ***********************/

/* side length computer
 *
 * precomputes the length of each side.
 * THREADS: num_sides
 */ 
__global__ void preval_side_length(double *s_length, 
                              double *s_V1x, double *s_V1y, 
                              double *s_V2x, double *s_V2y,
                              int num_sides) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < num_sides) {
        // compute and store the length of the side
        s_length[idx] = sqrtf(pow(s_V1x[idx] - s_V2x[idx],2) + pow(s_V1y[idx] - s_V2y[idx],2));
    }
}

/* jacobian computing
 *
 * precomputes the jacobian determinant for each element.
 * THREADS: num_elem
 */
__global__ void preval_jacobian(double *J, 
                           double *V1x, double *V1y, 
                           double *V2x, double *V2y, 
                           double *V3x, double *V3y,
                           int num_elem) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < num_elem) {
        double x1, y1, x2, y2, x3, y3;

        // read vertex points
        x1 = V1x[idx];
        y1 = V1y[idx];
        x2 = V2x[idx];
        y2 = V2y[idx];
        x3 = V3x[idx];
        y3 = V3y[idx];

        // calculate jacobian determinant
        // x = x2 * r + x3 * s + x1 * (1 - r - s)
        J[idx] = (x2 - x1) * (y3 - y1) - (x3 - x1) * (y2 - y1);
    }
}

/* evaluate normal vectors
 *
 * computes the normal vectors for each element along each side.
 * THREADS: num_sides
 *
 */
__global__ void preval_normals(double *Nx, double *Ny, 
                          double *s_V1x, double *s_V1y, 
                          double *s_V2x, double *s_V2y,
                          double *V1x, double *V1y, 
                          double *V2x, double *V2y, 
                          double *V3x, double *V3y,
                          int *left_side_number, int num_sides) {

    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < num_sides) {
        double x, y, length;
        double sv1x, sv1y, sv2x, sv2y;
    
        sv1x = s_V1x[idx];
        sv1y = s_V1y[idx];
        sv2x = s_V2x[idx];
        sv2y = s_V2y[idx];
    
        // lengths of the vector components
        x = sv2x - sv1x;
        y = sv2y - sv1y;
    
        // normalize
        length = sqrtf(pow(x, 2) + pow(y, 2));

        // store the result
        Nx[idx] = -y / length;
        Ny[idx] =  x / length;
    }
}

__global__ void preval_normals_direction(double *Nx, double *Ny, 
                          double *V1x, double *V1y, 
                          double *V2x, double *V2y, 
                          double *V3x, double *V3y,
                          int *left_elem, int *left_side_number, int num_sides) {

    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < num_sides) {
        double new_x, new_y, dot;
        double initial_x, initial_y, target_x, target_y;
        double x, y;
        int left_idx, side;

        // get left side's vertices
        left_idx = left_elem[idx];
        side     = left_side_number[idx];

        // get the normal vector
        x = Nx[idx];
        y = Ny[idx];
    
        // make it point the correct direction by learning the third vertex point
        switch (side) {
            case 0: 
                target_x = V3x[left_idx];
                target_y = V3y[left_idx];
                initial_x = (V1x[left_idx] + V2x[left_idx]) / 2.;
                initial_y = (V1y[left_idx] + V2y[left_idx]) / 2.;
                break;
            case 1:
                target_x = V1x[left_idx];
                target_y = V1y[left_idx];
                initial_x = (V2x[left_idx] + V3x[left_idx]) / 2.;
                initial_y = (V2y[left_idx] + V3y[left_idx]) / 2.;
                break;
            case 2:
                target_x = V2x[left_idx];
                target_y = V2y[left_idx];
                initial_x = (V1x[left_idx] + V3x[left_idx]) / 2.;
                initial_y = (V1y[left_idx] + V3y[left_idx]) / 2.;
                break;
        }

        // create the vector pointing towards the third vertex point
        new_x = target_x - initial_x;
        new_y = target_y - initial_y;

        // find the dot product between the normal and new vectors
        dot = x * new_x + y * new_y;
        
        if (dot > 0) {
            Nx[idx] *= -1;
            Ny[idx] *= -1;
        }
    }
}

__global__ void preval_partials(double *V1x, double *V1y,
                                double *V2x, double *V2y,
                                double *V3x, double *V3y,
                                double *xr,  double *yr,
                                double *xs,  double *ys, int num_elem) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < num_elem) {
        // evaulate the jacobians of the mappings for the chain rule
        // x = x2 * r + x3 * s + x1 * (1 - r - s)
        xr[idx] = V2x[idx] - V1x[idx];
        yr[idx] = V2y[idx] - V1y[idx];
        xs[idx] = V3x[idx] - V1x[idx];
        ys[idx] = V3y[idx] - V1y[idx];
    }
}

/***********************
 *
 * MAIN FUNCTIONS
 *
 ***********************/

/* riemann evaluation
 *
 * device function to solve the riemann problem.
 */
__device__ void eval_left_right(double *c_rho_left, double *c_rho_right,
                             double *c_u_left,   double *c_u_right,
                             double *c_v_left,   double *c_v_right,
                             double *c_E_left,   double *c_E_right,
                             double *rho_left, double *u_left, double *v_left, double *E_left,
                             double *rho_right, double *u_right, double *v_right, double *E_right,
                             double v1x, double v1y,
                             double v2x, double v2y,
                             double v3x, double v3y,
                             int j, // j, as usual, is the index of the integration point
                             int left_side, int right_side,
                             int left_idx, int right_idx,
                             int n_p, int n_quad1d,
                             int num_sides, double t) { 

    int i;

    // evaluate rho, u, v, E at the integration points
    *rho_left = 0.;
    *u_left   = 0.;
    *v_left   = 0.;
    *E_left   = 0.;
    *rho_right = 0.;
    *u_right   = 0.;
    *v_right   = 0.;
    *E_right   = 0.;
    
    for (i = 0; i < n_p; i++) {
        *rho_left += c_rho_left[i] * basis_side[left_side * n_p * n_quad1d + i * n_quad1d + j];
        *u_left   += c_u_left[i]   * basis_side[left_side * n_p * n_quad1d + i * n_quad1d + j];
        *v_left   += c_v_left[i]   * basis_side[left_side * n_p * n_quad1d + i * n_quad1d + j];
        *E_left   += c_E_left[i]   * basis_side[left_side * n_p * n_quad1d + i * n_quad1d + j];
    }

    // since we actually have coefficients for rho * u and rho * v
    *u_left = *u_left / *rho_left;
    *v_left = *v_left / *rho_left;

    // make all threads in the first warps be boundary sides
    if (right_idx == -1) {
        double r1_eval, r2_eval;
        double x, y;

        // we need the mapping back to the grid space
        switch (left_side) {
            case 0: 
                r1_eval = 0.5 + 0.5 * r_oned[j];
                r2_eval = 0.;
                break;
            case 1: 
                r1_eval = (1. - r_oned[j]) / 2.;
                r2_eval = (1. + r_oned[j]) / 2.;
                break;
            case 2: 
                r1_eval = 0.;
                r2_eval = 0.5 + 0.5 * r_oned[n_quad1d - 1 - j];
                break;
        }

        // x = x2 * r + x3 * s + x1 * (1 - r - s)
        x = v2x * r1_eval + v3x * r2_eval + v1x * (1 - r1_eval - r2_eval);
        y = v2y * r1_eval + v3y * r2_eval + v1y * (1 - r1_eval - r2_eval);
            
        // deal with the boundary element here
        *rho_right = *rho_left;
        *u_right   = *u_left;
        *v_right   = *v_left;
        *E_right   = *E_left;
        //*rho_right = boundary_exact_rho(x, y, t);
        //*u_right   = boundary_exact_u(x, y, t);
        //*v_right   = boundary_exact_v(x, y, t);
        //*E_right   = boundary_exact_E(x, y, t);

    } else {
        // evaluate the right side at the integration point
        for (i = 0; i < n_p; i++) {
            *rho_right += c_rho_right[i] * basis_side[right_side * n_p * n_quad1d + i * n_quad1d + n_quad1d - 1 - j];
            *u_right   += c_u_right[i]   * basis_side[right_side * n_p * n_quad1d + i * n_quad1d + n_quad1d - 1 - j];
            *v_right   += c_v_right[i]   * basis_side[right_side * n_p * n_quad1d + i * n_quad1d + n_quad1d - 1 - j];
            *E_right   += c_E_right[i]   * basis_side[right_side * n_p * n_quad1d + i * n_quad1d + n_quad1d - 1 - j];
        }

        // again, since we have coefficients for rho * u and rho * v
        *u_right = *u_right / *rho_right;
        *v_right = *v_right / *rho_right;
    }
}

/* surface integral evaluation
 *
 * evaluate all the riemann problems for each element.
 * THREADS: num_sides
 */
/*
 * d_t [   rho   ] + d_x [     rho * u    ] + d_y [    rho * v     ] = 0
 * d_t [ rho * u ] + d_x [ rho * u^2 + p  ] + d_y [   rho * u * v  ] = 0
 * d_t [ rho * v ] + d_x [  rho * u * v   ] + d_y [  rho * v^2 + p ] = 0
 * d_t [    E    ] + d_x [ u * ( E +  p ) ] + d_y [ v * ( E +  p ) ] = 0
 */

/* evaluate c
 *
 * evaulates the speed of sound c
 */
__device__ double eval_c(double rho, double u, double v, double E) {
    return sqrtf(GAMMA * pressure(rho, u, v, E) / rho);
}    

/* evaluate lambda
 *
 * finds the max absolute value of the jacobian for F(u).
 *  |u - c|, |u|, |u + c|
 */
__device__ double eval_lambda(double *c_rho_left, double *c_rho_right,
                             double *c_u_left,   double *c_u_right,
                             double *c_v_left,   double *c_v_right,
                             double *c_E_left,   double *c_E_right,
                             double J, int n_p, int n_quad) {
    double sum1_l, sum2_l, sum3_l;
    double sum1_r, sum2_r, sum3_r;
    double max;
    double rho, u, v, E, c;
    int i, j;

    ////////////////
    // left element 
    ////////////////

    // evaluate u - c
    sum1_l = 0;
    for (j = 0; j < n_quad; j++) {
        // evaluate rho,u,v,E at the integration point
        rho = 0.;
        u   = 0.;
        v   = 0.;
        E   = 0.;
        for (i = 0; i < n_p; i++) {
            rho += c_rho_left[i] * basis[n_quad * i + j];
            u   += c_u_left[i]   * basis[n_quad * i + j];
            v   += c_v_left[i]   * basis[n_quad * i + j];
            E   += c_E_left[i]   * basis[n_quad * i + j];
        }
        u = u / rho;
        v = v / rho;
        // evaluate c at the integration point
        c = eval_c(rho, u, v, E);

        sum1_l += w[j] * (u - c);
    }
    sum1_l = abs(sum1_l);

    // evaluate u
    sum2_l = 0;
    for (j = 0; j < n_quad; j++) {
        // evaluate u at the integration point
        u   = 0.;
        for (i = 0; i < n_p; i++) {
            u   += c_u_left[i]   * basis[n_quad * i + j];
        }
        u = u / rho;

        sum2_l += w[j] * u;
    }

    sum2_l = abs(sum2_l);

    // evaluate u + c
    sum3_l = 0;
    for (j = 0; j < n_quad; j++) {
        // evaluate rho,u,v,E at the integration point
        rho = 0.;
        u   = 0.;
        v   = 0.;
        E   = 0.;
        for (i = 0; i < n_p; i++) {
            rho += c_rho_left[i] * basis[n_quad * i + j];
            u   += c_u_left[i]   * basis[n_quad * i + j];
            v   += c_v_left[i]   * basis[n_quad * i + j];
            E   += c_E_left[i]   * basis[n_quad * i + j];
        }
        u = u / rho;
        v = v / rho;
        // evaluate c at the integration point
        c = eval_c(rho, u, v, E);

        sum3_l += w[j] * (u + c);
    }
    sum3_l = abs(sum3_l);

    ////////////////
    // right element
    ////////////////

    // evaluate u - c
    sum1_r = 0;
    for (j = 0; j < n_quad; j++) {
        // evaluate rho,u,v,E at the integration point
        rho = 0.;
        u   = 0.;
        v   = 0.;
        E   = 0.;
        for (i = 0; i < n_p; i++) {
            rho += c_rho_right[i] * basis[n_quad * i + j];
            u   += c_u_right[i]   * basis[n_quad * i + j];
            v   += c_v_right[i]   * basis[n_quad * i + j];
            E   += c_E_right[i]   * basis[n_quad * i + j];
        }
        u = u / rho;
        v = v / rho;
        // evaluate c at the integration point
        c = eval_c(rho, u, v, E);

        sum1_r += w[j] * (u - c);
    }
    sum1_r = abs(sum1_r);

    // evaluate u
    sum2_r = 0;
    for (j = 0; j < n_quad; j++) {
        // evaluate u at the integration point
        u   = 0.;
        for (i = 0; i < n_p; i++) {
            u   += c_u_right[i]   * basis[n_quad * i + j];
        }
        u = u / rho;

        sum2_r += w[j] * u;
    }

    sum2_r = abs(sum2_r);

    // evaluate u + c
    sum3_r = 0;
    for (j = 0; j < n_quad; j++) {
        // evaluate rho,u,v,E at the integration point
        rho = 0.;
        u   = 0.;
        v   = 0.;
        E   = 0.;
        for (i = 0; i < n_p; i++) {
            rho += c_rho_right[i] * basis[n_quad * i + j];
            u   += c_u_right[i]   * basis[n_quad * i + j];
            v   += c_v_right[i]   * basis[n_quad * i + j];
            E   += c_E_right[i]   * basis[n_quad * i + j];
        }
        u = u / rho;
        v = v / rho;
        // evaluate c at the integration point
        c = eval_c(rho, u, v, E);

        sum3_r += w[j] * (u + c);
    }
    sum3_r = abs(sum3_r);

    max = 0;
    if (sum1_l > max) {
        max = sum1_l;
    }
    if (sum2_l > max) {
        max = sum2_l;
    }
    if (sum3_l > max) {
        max = sum3_l;
    }
    if (sum1_r > max) {
        max = sum1_r;
    }
    if (sum2_r > max) {
        max = sum2_r;
    }
    if (sum3_r > max) {
        max = sum3_r;
    }

    return max / 2 * J;
}

/* evaluate flux
 *
 * takes the actual values of rho, u, v, and E and returns the flux 
 * x and y components. 
 * NOTE: this needs the ACTUAL values for u and v, NOT rho * u, rho * v.
 */
__device__ void eval_flux(double rho, double u, double v, double E, 
                     double *flux_x1, double *flux_y1,
                     double *flux_x2, double *flux_y2,
                     double *flux_x3, double *flux_y3,
                     double *flux_x4, double *flux_y4) {

    // evaluate pressure
    double p = pressure(rho, u, v, E);

    // flux_1 
    *flux_x1 = rho * u;
    *flux_y1 = rho * v;

    // flux_2
    *flux_x2 = rho * u * u + p;
    *flux_y2 = rho * u * v;

    // flux_3
    *flux_x3 = rho * u * v;
    *flux_y3 = rho * v * v + p;

    // flux_4
    *flux_x4 = u * (E + p);
    *flux_y4 = v * (E + p);
}

__device__ void eval_surface(double *c_rho_left, double *c_u_left, double *c_v_left, double *c_E_left,
                             double *c_rho_right, double *c_u_right, double *c_v_right, double *c_E_right,
                             double *left_riemann_rhs, double *right_riemann_rhs, 
                             double len, double J,
                             double v1x, double v1y,
                             double v2x, double v2y,
                             double v3x, double v3y,
                             int left_idx,  int right_idx,
                             int left_side, int right_side, 
                             double nx, double ny, 
                             int n_quad1d, int n_quad, int n_p, int num_sides, 
                             int num_elem, double t, int idx) {
    int i, j;
    double s;
    double lambda;
    double left_sum1, right_sum1;
    double left_sum2, right_sum2;
    double left_sum3, right_sum3;
    double left_sum4, right_sum4;
    double flux_x1_l, flux_x2_l, flux_x3_l, flux_x4_l;
    double flux_x1_r, flux_x2_r, flux_x3_r, flux_x4_r;
    double flux_y1_l, flux_y2_l, flux_y3_l, flux_y4_l;
    double flux_y1_r, flux_y2_r, flux_y3_r, flux_y4_r;
    double rho_left, u_left, v_left, E_left;
    double rho_right, u_right, v_right, E_right;

    // multiply across by the i'th basis function
    for (i = 0; i < n_p; i++) {

        left_sum1  = 0.;
        left_sum2  = 0.;
        left_sum3  = 0.;
        left_sum4  = 0.;
        right_sum1 = 0.;
        right_sum2 = 0.;
        right_sum3 = 0.;
        right_sum4 = 0.;

        for (j = 0; j < n_quad1d; j++) {

            // calculate the left and right values along the surface
            eval_left_right(c_rho_left, c_rho_right,
                            c_u_left,   c_u_right,
                            c_v_left,   c_v_right,
                            c_E_left,   c_E_right,
                            &rho_left, &u_left, &v_left, &E_left,
                            &rho_right, &u_right, &v_right, &E_right,
                            v1x, v1y, v2x, v2y, v3x, v3y,
                            j, left_side, right_side,
                            left_idx, right_idx,
                            n_p, n_quad1d, num_sides, t);

            // calculate the left fluxes
            eval_flux(rho_left, u_left, v_left, E_left,
                      &flux_x1_l, &flux_y1_l, &flux_x2_l, &flux_y2_l,
                      &flux_x3_l, &flux_y3_l, &flux_x4_l, &flux_y4_l);

            // calculate the right fluxes
            eval_flux(rho_right, u_right, v_right, E_right,
                      &flux_x1_r, &flux_y1_r, &flux_x2_r, &flux_y2_r,
                      &flux_x3_r, &flux_y3_r, &flux_x4_r, &flux_y4_r);

            // need these local max values
            lambda = eval_lambda(c_rho_left, c_rho_right,
                                 c_u_left, c_u_right, 
                                 c_v_left, c_v_right, 
                                 c_E_left, c_E_right,
                                 J, n_p, n_quad);
            
            // 1st equation
            s = 0.5 * ((flux_x1_l + flux_x1_r) * nx + (flux_y1_l + flux_y1_r) * ny 
                        + lambda * (rho_left - rho_right));
            left_sum1  += w_oned[j] * s * basis_side[left_side * n_p * n_quad1d + i * n_quad1d + j];
            right_sum1 += w_oned[j] * s * basis_side[right_side * n_p * n_quad1d + i * n_quad1d + n_quad1d - 1 - j];

            // 2nd equation
            s = 0.5 * ((flux_x2_l + flux_x2_r) * nx + (flux_y2_l + flux_y2_r) * ny 
                        + lambda * (u_left - u_right));
            left_sum2  += w_oned[j] * s * basis_side[left_side * n_p * n_quad1d + i * n_quad1d + j];
            right_sum2 += w_oned[j] * s * basis_side[right_side * n_p * n_quad1d + i * n_quad1d + n_quad1d - 1 - j];

            // 3rd equation
            s = 0.5 * ((flux_x3_l + flux_x3_r) * nx + (flux_y3_l + flux_y3_r) * ny 
                        + lambda * (v_left - v_right));
            left_sum3  += w_oned[j] * s * basis_side[left_side * n_p * n_quad1d + i * n_quad1d + j];
            right_sum3 += w_oned[j] * s * basis_side[right_side * n_p * n_quad1d + i * n_quad1d + n_quad1d - 1 - j];

            // 4th equation
            s = 0.5 * ((flux_x4_l + flux_x4_r) * nx + (flux_y4_l + flux_y4_r) * ny 
                        + lambda * (E_left - E_right));
            left_sum4  += w_oned[j] * s * basis_side[left_side * n_p * n_quad1d + i * n_quad1d + j];
            right_sum4 += w_oned[j] * s * basis_side[right_side * n_p * n_quad1d + i * n_quad1d + n_quad1d - 1 - j];
        }

        // store this side's contribution in the riemann rhs vectors
        left_riemann_rhs[num_sides * n_p * 0 + i * num_sides + idx]  = -len / 2 * left_sum1;
        left_riemann_rhs[num_sides * n_p * 1 + i * num_sides + idx]  = -len / 2 * left_sum2;
        left_riemann_rhs[num_sides * n_p * 2 + i * num_sides + idx]  = -len / 2 * left_sum3;
        left_riemann_rhs[num_sides * n_p * 3 + i * num_sides + idx]  = -len / 2 * left_sum4;
        right_riemann_rhs[num_sides * n_p * 0 + i * num_sides + idx] =  len / 2 * right_sum1;
        right_riemann_rhs[num_sides * n_p * 1 + i * num_sides + idx] =  len / 2 * right_sum2;
        right_riemann_rhs[num_sides * n_p * 2 + i * num_sides + idx] =  len / 2 * right_sum3;
        right_riemann_rhs[num_sides * n_p * 3 + i * num_sides + idx] =  len / 2 * right_sum4;
    }
}

/* volume integrals
 *
 * evaluates and adds the volume integral to the rhs vector
 * THREADS: num_elem
 */
__device__ void eval_volume(double *c_rho, double *c_u,
                            double *c_v,   double *c_E,
                            double *quad_rhs, 
                            double x_r, double y_r,
                            double x_s, double y_s,
                            int n_quad, int n_p, int num_elem, int idx) {
    int i, j, k;
    double rho, u, v, E;
    double flux_x1, flux_y1, flux_x2, flux_y2;
    double flux_x3, flux_y3, flux_x4, flux_y4;
    double sum1, sum2, sum3, sum4;

    // evaluate the volume integral for each coefficient
    for (i = 0; i < n_p; i++) {
        sum1 = 0.;
        sum2 = 0.;
        sum3 = 0.;
        sum4 = 0.;
        for (j = 0; j < n_quad; j++) {

            // evaluate rho, u, v, E at the integration point.
            rho = 0.;
            u   = 0.;
            v   = 0.;
            E   = 0.;
            for (k = 0; k < n_p; k++) {
                rho += c_rho[k] * basis[n_quad * k + j];
                u   += c_u[k]   * basis[n_quad * k + j];
                v   += c_v[k]   * basis[n_quad * k + j];
                E   += c_E[k]   * basis[n_quad * k + j];
            }

            // since we actually have coefficients for rho * u, rho * v
            u = u / rho;
            v = v / rho;

            // evaluate flux
            eval_flux(rho, u, v, E,
                 &flux_x1, &flux_y1, &flux_x2, &flux_y2,
                 &flux_x3, &flux_y3, &flux_x4, &flux_y4);
                 
            // Add to the sum
            // [fx fy] * [y_s, -y_r; -x_s, x_r] * [phi_x phi_y]
            sum1 += (  flux_x1 * ( basis_grad_x[n_quad * i + j] * y_s
                                  -basis_grad_y[n_quad * i + j] * y_r)
                     + flux_y1 * (-basis_grad_x[n_quad * i + j] * x_s 
                                 + basis_grad_y[n_quad * i + j] * x_r));
            sum2 += (  flux_x2 * ( basis_grad_x[n_quad * i + j] * y_s
                                  -basis_grad_y[n_quad * i + j] * y_r)
                     + flux_y2 * (-basis_grad_x[n_quad * i + j] * x_s 
                                 + basis_grad_y[n_quad * i + j] * x_r));
            sum3 += (  flux_x3 * ( basis_grad_x[n_quad * i + j] * y_s
                                  -basis_grad_y[n_quad * i + j] * y_r)
                     + flux_y3 * (-basis_grad_x[n_quad * i + j] * x_s 
                                 + basis_grad_y[n_quad * i + j] * x_r));
            sum4 += (  flux_x4 * ( basis_grad_x[n_quad * i + j] * y_s
                                  -basis_grad_y[n_quad * i + j] * y_r)
                     + flux_y4 * (-basis_grad_x[n_quad * i + j] * x_s 
                                 + basis_grad_y[n_quad * i + j] * x_r));
        }

        // store the result
        quad_rhs[num_elem * n_p * 0 + i * num_elem + idx] = sum1;
        quad_rhs[num_elem * n_p * 1 + i * num_elem + idx] = sum2;
        quad_rhs[num_elem * n_p * 2 + i * num_elem + idx] = sum3;
        quad_rhs[num_elem * n_p * 3 + i * num_elem + idx] = sum4;
    }
}

/* evaluate error
 * 
 * evaluates u at the three vertex points for output
 * THREADS: num_elem
 */
__device__ void eval_error(double *c, 
                       double v1x, double v1y,
                       double v2x, double v2y,
                       double v3x, double v3y,
                       double *Uv1, double *Uv2, double *Uv3,
                       int num_elem, int n_p, double t, int idx) {

    int i;
    double uv1, uv2, uv3;

    // calculate values at three vertex points
    uv1 = 0.;
    uv2 = 0.;
    uv3 = 0.;
    for (i = 0; i < n_p; i++) {
        uv1 += c[i] * basis_vertex[i * 3 + 0];
        uv2 += c[i] * basis_vertex[i * 3 + 1];
        uv3 += c[i] * basis_vertex[i * 3 + 2];
    }

    // store result
    Uv1[idx] = uv1 - uexact(v1x, v1y, t);
    Uv2[idx] = uv2 - uexact(v2x, v2y, t);
    Uv3[idx] = uv3 - uexact(v3x, v3y, t);
}

/* evaluate u
 * 
 * evaluates rho and E at the three vertex points for output
 * THREADS: num_elem
 */
__device__ void eval_u(double *c, 
                       double *Uv1, double *Uv2, double *Uv3,
                       int num_elem, int n_p, int idx) {
    int i;
    double uv1, uv2, uv3;

    // calculate values at the integration points
    uv1 = 0.;
    uv2 = 0.;
    uv3 = 0.;
    for (i = 0; i < n_p; i++) {
        uv1 += c[i] * basis_vertex[i * 3 + 0];
        uv2 += c[i] * basis_vertex[i * 3 + 1];
        uv3 += c[i] * basis_vertex[i * 3 + 2];
    }

    // store result
    Uv1[idx] = uv1;
    Uv2[idx] = uv2;
    Uv3[idx] = uv3;
}
/* evaluate u velocity
 * 
 * evaluates u and v at the three vertex points for output
 * THREADS: num_elem
 */
__device__ void eval_u_velocity(double *c, double *c_rho,
                       double *Uv1, double *Uv2, double *Uv3,
                       int num_elem, int n_p, int idx) {
    int i;
    double uv1, uv2, uv3;
    double rhov1, rhov2, rhov3;

    // calculate values at the integration points
    rhov1 = 0.;
    rhov2 = 0.;
    rhov3 = 0.;
    for (i = 0; i < n_p; i++) {
        rhov1 += c_rho[i] * basis_vertex[i * 3 + 0];
        rhov2 += c_rho[i] * basis_vertex[i * 3 + 1];
        rhov3 += c_rho[i] * basis_vertex[i * 3 + 2];
    }

    uv1 = 0.;
    uv2 = 0.;
    uv3 = 0.;
    for (i = 0; i < n_p; i++) {
        uv1 += c[i] * basis_vertex[i * 3 + 0];
        uv2 += c[i] * basis_vertex[i * 3 + 1];
        uv3 += c[i] * basis_vertex[i * 3 + 2];
    }

    uv1 = uv1 / rhov1;
    uv2 = uv2 / rhov2;
    uv3 = uv3 / rhov3;

    // store result
    Uv1[idx] = uv1;
    Uv2[idx] = uv2;
    Uv3[idx] = uv3;
}
